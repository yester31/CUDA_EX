#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "sgemm0.h"

#define CEIL_DIV(M, N) (((M) + (N) - 1) / (N))

using half = __half;

// kernel program for the device (GPU): compiled by NVCC
template <typename T>
__global__ void sgemm_kernel_navie(
    T const *A,
    T const *B,
    T *C,
    int32_t const M,
    int32_t const K,
    int32_t const N,
    T const alpha,
    T const beta)
{
    // compute position in C that this thread is responsible for
    const uint row = blockIdx.x * blockDim.x + threadIdx.x;
    const uint col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row >= M || col >= N)
        return;

    T sum = 0;
    for (int k = 0; k < K; ++k)
    {
        sum += A[row * K + k] * B[k * N + col];
    }
    C[row * N + col] = alpha * sum + beta * C[row * N + col];
}

template <typename T>
hipError_t SGEMM_Naive_Impl(hipStream_t stream,
                             T const *A,
                             T const *B,
                             T *C,
                             int32_t const M,
                             int32_t const K,
                             int32_t const N,
                             T const alpha,
                             T const beta)
{
    // row-> x, col-> y
    // launch a kernel on the GPU with one thread for each element.
    dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32), 1);
    // 32 * 32 = 1024 thread per block
    dim3 blockDim(32, 32, 1);

    sgemm_kernel_navie<T><<<gridDim, blockDim, 0, stream>>>(A, B, C, M, K, N, alpha, beta);
    return hipGetLastError();
}

#define SPECIALIZED_IMPL(T) \
    template hipError_t SGEMM_Naive_Impl<T>(hipStream_t stream, T const *A, T const *B, T *C, int32_t const M, int32_t const K, int32_t const N, T const alpha, T const beta);

SPECIALIZED_IMPL(float)
SPECIALIZED_IMPL(half)