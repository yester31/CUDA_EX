#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "sgemm.h"

using half = __half;

// kernel program for the device (GPU): compiled by NVCC
template <typename T>
__global__ void sgemm_kernel_navie(
    T const *A,
    T const *B,
    T *C,
    int32_t const M,
    int32_t const K,
    int32_t const N,
    T const alpha,
    T const beta,
    const int tcount)
{
    int pos = threadIdx.x + blockIdx.x * blockDim.x;
    if (pos >= tcount)
        return;

    int x_idx = pos % N;
    int y_idx = pos / N;
    T sum = 0;
    for (int k = 0; k < K; ++k)
    {
        sum += A[y_idx * K + k] * B[k * N + x_idx];
    }
    C[y_idx * N + x_idx] = alpha * sum + beta * C[y_idx * N + x_idx];
}

template <typename T>
__global__ void PreprocForward(T *output,      // [N,C(RGB),H,W]
                               T const *input, // [N,H,W,C(BGR)]
                               int32_t const batchSize, int32_t const channel, int32_t const height, int32_t const width,
                               int32_t const nthreads) // nthreads
{
    size_t pos = threadIdx.x + blockIdx.x * blockDim.x;
    if (pos >= nthreads)
        return;

    const int32_t w_idx = pos % width;
    int32_t idx = pos / width;
    const int32_t h_idx = idx % height;
    idx /= height;
    const int32_t c_idx = idx % channel;
    const int32_t b_idx = idx / channel;

    int32_t s_idx = b_idx * height * width * channel + h_idx * width * channel + w_idx * channel + (channel - 1) - c_idx;

    output[pos] = input[s_idx] / static_cast<T>(255.);
}

template <typename T>
hipError_t SGEMM_Naive_Impl(hipStream_t stream,
                             T const *A,
                             T const *B,
                             T *C,
                             int32_t const M,
                             int32_t const K,
                             int32_t const N,
                             T const alpha,
                             T const beta)
{
    // launch a kernel on the GPU with one thread for each element.
    int thread_cnt = M * N;
    int block = 256;
    int grid = (thread_cnt - 1) / block + 1;

    dim3 dimGrid(grid, 1, 1);
    dim3 dimBlock(block, 1, 1); // x,y,z
    sgemm_kernel_navie<T><<<dimGrid, dimBlock, 0, stream>>>(A, B, C, M, K, N, alpha, beta, thread_cnt);

    return hipGetLastError();
}

#define SPECIALIZED_IMPL(T) \
    template hipError_t SGEMM_Naive_Impl<T>(hipStream_t stream, T const *A, T const *B, T *C, int32_t const M, int32_t const K, int32_t const N, T const alpha, T const beta);

SPECIALIZED_IMPL(float)
SPECIALIZED_IMPL(half)