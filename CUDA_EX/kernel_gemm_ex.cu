#include "hip/hip_runtime.h"
#include "util_cuda.cuh"

//kernel program for the device (GPU): compiled by NVCC
__global__ void matrixMulKernel_2d(
	float* output, const float* input_a, const float* input_b,
	int M, int K, int N, const int tcount)
{
	int pos = threadIdx.x + blockIdx.x * blockDim.x;
	if (pos >= tcount) return;
	
	int w_idx = pos % N;
	int h_idx = pos / N;
	int sum = 0;
	for (int k = 0; k < K; ++k) {
		sum += input_a[h_idx * K + k] * input_b[k * N + w_idx];
	}
	output[h_idx * N + w_idx] = sum;
}


int main(void) {
	// A[M, K] * B[K, N] = C[M, N]
	const int M = 128;
	const int K = 256;
	const int N = 128;

	std::vector<float> input_a(M * K);
	std::vector<float> input_b(K * N);
	std::vector<float> output(M * N);
	std::vector<float> output_cpu(M * N);

	// input data �ʱ�ȭ
	generate_data_f(input_a.data(), input_a.size());
	generate_data_f(input_b.data(), input_b.size());

	//device-side data
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_o = 0;

	// allocate device memory
	CUDA_CHECK(hipMalloc((void**)&dev_a, input_a.size() * sizeof(float)));
	CUDA_CHECK(hipMalloc((void**)&dev_b, input_b.size() * sizeof(float)));
	CUDA_CHECK(hipMalloc((void**)&dev_o, output.size() * sizeof(float)));

	uint64_t start_time1 = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

	//copy from host to device 
	CUDA_CHECK(hipMemcpy(dev_a, input_a.data(), input_a.size() * sizeof(float), hipMemcpyHostToDevice));//dev_a=a;
	CUDA_CHECK(hipMemcpy(dev_b, input_b.data(), input_b.size() * sizeof(float), hipMemcpyHostToDevice));//dev_b=b;

	//launch a kernel on the GPU with one thread for each element.
	int thread_cnt = output.size();
	int block = 256;
	int grid = (thread_cnt - 1) / block + 1;

	dim3 dimGrid(grid, 1, 1);
	dim3 dimBlock(block, 1, 1);//x,y,z

	uint64_t start_time2 = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

	matrixMulKernel_2d << <dimGrid, dimBlock >> > (dev_o, dev_a, dev_b, M, K, N, thread_cnt);
	CUDA_CHECK(hipPeekAtLastError());

	uint64_t start_time3 = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

	//copy from device to host
	CUDA_CHECK(hipMemcpy(output.data(), dev_o, output.size() * sizeof(float), hipMemcpyDeviceToHost));//c=dev_c;

	uint64_t start_time4 = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

	//free device memory
	CUDA_CHECK(hipFree(dev_o));
	CUDA_CHECK(hipFree(dev_a));
	CUDA_CHECK(hipFree(dev_b));

	uint64_t start_time5 = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
	
	//validate gpu kernel function
	for (int m = 0; m < M; ++m) {
		for (int n = 0; n < N; ++n) {
			float sum = 0.f;
			for (int k = 0; k < K; ++k) {
				sum += input_a[m * K + k] * input_b[k * N + n];
			}
			output_cpu[m * N + n] = sum;
		} 
	}
	uint64_t start_time6 = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now().time_since_epoch()).count();

	// ��� ����
	valid_results_f(output, output_cpu);

	printf("dur_time(gpu) w = %6.3f [msec] \n",	(start_time4 - start_time1) / 1000.f);
	printf("dur_time(gpu) wo = %6.3f [msec] \n",(start_time3 - start_time2) / 1000.f);
	printf("dur_time(cpu) = %6.3f [msec] \n",	(start_time6 - start_time5) / 1000.f);

	return 0;
}