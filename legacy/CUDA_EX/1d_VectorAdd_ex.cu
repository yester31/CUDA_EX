#include "hip/hip_runtime.h"
#include "util_cuda.cuh"

using namespace std;
using namespace chrono;
#define NUM_DATA 10240000
#define BLOCK_SIZE 1024

// kernel �Լ�
__global__ void vecAdd(int *_a, int *_b, int *_c) {
	int tID = blockIdx.x * blockDim.x + threadIdx.x;
	_c[tID] = _a[tID] + _b[tID];
}

int main(void) {

	// start the timer
	uint64_t total_time = 0;

	// ������ ���� ����
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;

	// �Ҵ��� �޸𸮰��� ������ ��� 
	int memSize = sizeof(int) * NUM_DATA;
	printf("%d elements, memSize = %d bytes\n", NUM_DATA, memSize);

	// �޸� ������ ������ �� �ּҸ� �޸� �����Ϳ� �Ҵ� �� �޸��� ������ 0���� memSize ũ�⸸ŭ ����
	a = new int[NUM_DATA]; memset(a, 0, memSize);
	b = new int[NUM_DATA]; memset(b, 0, memSize);
	c = new int[NUM_DATA]; memset(c, 0, memSize);

	// �Ҵ��� �޸� ������ ���꿡 ����� ������ �Ҵ�
	for (int i = 0; i < NUM_DATA; i++) {
		a[i] = rand() % 10;
		b[i] = rand() % 10;
	}

	// Device�� memSize ��ŭ�� ������ ���� �� ������ ������ Device���� �Ҵ�� ������ �ּҸ� ����
	hipMalloc(&d_a, memSize);
	hipMalloc(&d_b, memSize);
	hipMalloc(&d_c, memSize);

	uint64_t start_time1 = duration_cast<microseconds>(system_clock::now().time_since_epoch()).count();

	// Host -> Device ������ ���� (data transfer overhead)
	hipMemcpy(d_a, a, memSize, hipMemcpyHostToDevice); // ����� �۵���.
	hipMemcpy(d_b, b, memSize, hipMemcpyHostToDevice);

	dim3 block(BLOCK_SIZE);
	dim3 grid((NUM_DATA + block.x - 1) / block.x);

	uint64_t start_time2 = duration_cast<microseconds>(system_clock::now().time_since_epoch()).count();

	// cuda kernel �Լ� ȣ��
	vecAdd << <grid, block >> > (d_a, d_b, d_c);
	hipDeviceSynchronize();

	uint64_t start_time3 = duration_cast<microseconds>(system_clock::now().time_since_epoch()).count();

	// Device -> Host ������ ���� (data transfer overhead)
	hipMemcpy(c, d_c, memSize, hipMemcpyDeviceToHost);

	//end the timer
	uint64_t start_time4 = duration_cast<microseconds>(system_clock::now().time_since_epoch()).count();

	printf("dur_time(gpu) w = %6.3f [msec] \n", (start_time4 - start_time1) / 1000.f);
	printf("dur_time(gpu) wo = %6.3f [msec] \n", (start_time3 - start_time2) / 1000.f);

	// ��� ����
	bool result = true;
	for (int i = 0; i < NUM_DATA; i++) {
		if ((a[i] + b[i]) != c[i]) {
			printf("[%d] The results is not matched! (%d, %d)\n", i, a[i] + b[i], c[i]);
			result = false;
		}
	}

	if (result)
		printf("GPU works well! \n");

	uint64_t start_time5 = duration_cast<microseconds>(system_clock::now().time_since_epoch()).count();

	for (int i = 0; i < NUM_DATA; i++) {
		c[i] = a[i] + b[i];
	}
	//end the timer
	uint64_t start_time6 = duration_cast<microseconds>(system_clock::now().time_since_epoch()).count();
	printf("dur_time(cpu) = %6.3f [msec] \n", (start_time6 - start_time5) / 1000.f);

	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	delete[] a;	delete[] b;	delete[] c;
}